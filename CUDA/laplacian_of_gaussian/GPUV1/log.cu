#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>


__global__ void grayscale(unsigned char * data_rgb, unsigned char * data_gray, std::size_t rows, std::size_t cols)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if( i < cols && j < rows )
    {
        data_gray[ j * cols + i ] = ( 
                307 * data_rgb[ 3 * (j * cols + i) ]
            +   604 * data_rgb[ 3 * (j * cols + i) + 1 ]
            +   113 * data_rgb[ 3 * (j * cols + i) + 2 ]
        ) / 1024;
    }
}

__global__ void laplacian_of_gaussian(unsigned char const * const data_gray, unsigned char * const data_out, std::size_t rows, std::size_t cols)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;


    if( i > 2 && i < (cols - 2) && j > 2 && j < (rows - 2))
    {
        // Tous les pixels que l'on multiplie par 16
        auto result = data_gray[(j * cols + i)] * 16

        // Tous les pixels que l'on multiplie par -2
        + ( data_gray[((j-1) * cols + i)] + data_gray[((j+1) * cols + i)] + data_gray[(j * cols + (i-1))] + data_gray[(j * cols + (i+1))] ) * -2

        // Tous les pixels que l'on multiplie par -1
        + ( data_gray[((j-2) * cols + i)] + data_gray[((j+2) * cols + i)] + data_gray[(j * cols + (i-2))] + data_gray[(j * cols + (i+2))] 
            + data_gray[((j-1) * cols + (i-1))] + data_gray[((j-1) * cols + (i+1))] + data_gray[((j+1) * cols + (i-1))] + data_gray[((j+1) * cols + (i+1))] ) * -1;


        result = result * result;
        result = result > 255*255 ? result = 255*255 : result;
        data_out[ j * cols + i ] = sqrt((float) result);
    }
}

int main(int argc, char** argv)
{
    //printf("Number of argument : %d\n", argc);

    if(argc >= 2){

        int threadSize = 32;

        if(argc == 3){
            threadSize = atoi(argv[2]);
        }

        // Mesure de temps
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Récupère l'image
        cv::Mat image_in = cv::imread(argv[1], cv::IMREAD_UNCHANGED);

        // Récupère les informations des pixels
        auto data_rgb = image_in.data;
        auto rows = image_in.rows;
        auto cols = image_in.cols;

	
        std::cout << "rows = " << rows << " columns = " << cols << std::endl;

        // On crée les informations de sorties 
        std::vector<unsigned char> out(rows * cols); 
        // On crée l'image de sortie
        cv::Mat image_out(rows, cols, CV_8UC1, out.data());

        // On copie l'image d'entrée sur le device
        unsigned char * data_rgb_device;
        unsigned char * data_gray_device;
        // On crée une copie des informations de sortie sur le device
        unsigned char* data_out_device;

        hipMalloc(&data_rgb_device, 3 * rows * cols);  // 1 pixel = 3 couleurs
        hipMalloc(&data_gray_device, rows * cols);
        hipMalloc(&data_out_device, rows * cols);

        hipMemcpy(data_rgb_device, data_rgb,  3 * rows * cols, hipMemcpyHostToDevice );

        dim3 threads(threadSize, threadSize );
        dim3 blocks(( cols -1 ) / threads.x + 1 , ( rows - 1) / threads.y + 1);

        std::cout << "Nombre de threads = " << threads.x << "  " << threads.y << std::endl;
        std::cout << "Nombre de blocks = " << blocks.x << "  " << blocks.y << std::endl;

        // Lancement du timer
        hipEventRecord(start);
        
        grayscale<<< blocks , threads >>>(data_rgb_device, data_gray_device, rows, cols);

        // lancement du programme
        laplacian_of_gaussian<<< blocks , threads >>>(data_gray_device, data_out_device, rows, cols);

        // On arrête le timer
        hipEventRecord(stop);

        hipDeviceSynchronize();
        /*auto err = hipGetLastError();
        if( err != hipSuccess )
        {
            printf("Errors found :\n %s", hipGetErrorString(err));
        }*/

        // On copie les informations de sortie du device vers le host
        hipMemcpy(out.data(), data_out_device, rows * cols, hipMemcpyDeviceToHost );
        
        // On récupère le temps d'exécution
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Execution time : %f\n",milliseconds);

        cv::imwrite( "outCudaV1.jpg", image_out);

        // On libère l'espace sur le device
        hipFree(data_rgb_device);
        hipFree(data_gray_device);
        hipFree(data_out_device);
    }

    return 0;
}

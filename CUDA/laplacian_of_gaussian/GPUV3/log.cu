#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void laplacian_of_gaussian(unsigned char * data_rgb, unsigned char * const data_out, std::size_t rows, std::size_t cols)
{
    auto i = blockIdx.x * (blockDim.x - 4)+ threadIdx.x;
    auto j = blockIdx.y * (blockDim.y - 4) + threadIdx.y;

    auto gray_i = threadIdx.x;
    auto gray_j = threadIdx.y;

    extern __shared__ unsigned char data_gray[];

    auto cols_gray = blockDim.x;


    data_gray[ gray_j * cols_gray + gray_i ] = ( 
            307 * data_rgb[ 3 * (j * cols + i) ]
        +   604 * data_rgb[ 3 * (j * cols + i) + 1 ]
        +   113 * data_rgb[ 3 * (j * cols + i) + 2 ]
    ) / 1024;
        

    __syncthreads();


    if( gray_i > 1 && gray_i < (cols_gray - 2) && gray_j > 1 && gray_j < (blockDim.y - 2))
    {
        // Tous les pixels que l'on multiplie par 16
        auto result = data_gray[(gray_j * cols_gray + gray_i)] * 16

        // Tous les pixels que l'on multiplie par -2
        + ( data_gray[((gray_j-1) * cols_gray + gray_i)] + data_gray[((gray_j+1) * cols_gray + gray_i)] + data_gray[(gray_j * cols_gray + (gray_i-1))] + data_gray[(gray_j * cols_gray + (gray_i+1))] ) * -2

        // Tous les pixels que l'on multiplie par -1
        + ( data_gray[((gray_j-2) * cols_gray + gray_i)] + data_gray[((gray_j+2) * cols_gray + gray_i)] + data_gray[(gray_j * cols_gray + (gray_i-2))] + data_gray[(gray_j * cols_gray + (gray_i+2))] 
            + data_gray[((gray_j-1) * cols_gray + (gray_i-1))] + data_gray[((gray_j-1) * cols_gray + (gray_i+1))] + data_gray[((gray_j+1) * cols_gray + (gray_i-1))] + data_gray[((gray_j+1) * cols_gray + (gray_i+1))] ) * -1;


        result = result * result;
        result = result > 255*255 ? result = 255*255 : result;
        data_out[ j * cols + i ] = sqrt((float) result);
    }
}

int main(int argc, char** argv)
{
    printf("Number of argument : %d\n", argc);

    if(argc >= 2){

        hipError_t err;

        int threadSize = 32;
        int streamsNumber = 2;

        if(argc == 3){
            threadSize = atoi(argv[2]);
        }

        // Mesure de temps
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        //std::cout << "Création du timer faite" << std::endl;

        // Récupère l'image
        cv::Mat image_in = cv::imread(argv[1], cv::IMREAD_UNCHANGED);
        // Récupère les informations des pixels
        auto data_rgb = image_in.data;
        auto rows = image_in.rows;
        auto cols = image_in.cols;
        auto size_data_in = (rows  / streamsNumber) * cols * 3;
        auto size_data_out = (rows / streamsNumber)  * cols;

        unsigned char ** data_in_streams; 


        err = hipHostMalloc(&data_in_streams, streamsNumber);
        //if( err != hipSuccess ) { std::cerr << "Erreur malloc data_in_streams" << std::endl; } 

        for( std::size_t i = 0 ; i < streamsNumber ; ++i ){
            err = hipMalloc(&data_in_streams[i], size_data_in);
            //if( err != hipSuccess ) { std::cerr << "Erreur malloc data_in_streams[" << i << "]" << std::endl; } 
        }

        unsigned char ** data_out_streams; 


        err = hipHostMalloc(&data_out_streams, streamsNumber);
        //if( err != hipSuccess ) { std::cerr << "Erreur malloc data_out_streams" << std::endl; } 

        for( std::size_t i = 0 ; i < streamsNumber ; ++i ){
            err = hipMalloc(&data_out_streams[i], size_data_out);
            //if( err != hipSuccess ) { std::cerr << "Erreur malloc data_out_streams[" << i << "]" << std::endl; } 
        }

        //std::cout << "rows = " << rows << " columns = " << cols << std::endl;

        // On crée les informations de sorties 
        std::vector<unsigned char> out(rows * cols); 
        // On crée l'image de sortie
        cv::Mat image_out(rows, cols, CV_8UC1, out.data());

        //std::cout << "Image et données de sortie initialisées" << std::endl;

        //std::cout << "Image sur le device allouée" << std::endl;

        //std::cout << "Données de sortie sur le device allouées" << std::endl;

        hipStream_t streams[ streamsNumber ];

        for( std::size_t i = 0 ; i < streamsNumber ; ++i ){
            hipStreamCreate( &streams[i]);
        }

        for( std::size_t i = 0 ; i < streamsNumber ; ++i ){
            hipMemcpyAsync( data_in_streams[i], data_rgb + i * size_data_in, size_data_in, hipMemcpyHostToDevice, streams[i]);
        }
                                                                    
        //std::cout << "Image d'entrée mise sur le device" << std::endl;

        dim3 threads(threadSize, threadSize );
        dim3 blocks((( cols -1 ) / (threads.x-4)) + 1 , (( rows - 1) / (threads.y-4)) / streamsNumber + 1);

        //std::cout << "Nombre de threads = " << threads.x << "  " << threads.y << std::endl;
        //std::cout << "Nombre de blocks = " << blocks.x << "  " << blocks.y << std::endl;

        // Lancement du timer
        hipEventRecord(start);

        //std::cout << "Lancement du timer" << std::endl;
        
        for( std::size_t i = 0 ; i < streamsNumber ; ++i ){
            // lancement du programme
            laplacian_of_gaussian<<< blocks , threads , threadSize * threadSize, streams[i]>>>(data_in_streams[i], data_out_streams[i], rows, cols);
        }

        
        // On arrête le timer
        hipEventRecord(stop);

        //std::cout << "Fin du timer" << std::endl;

        hipDeviceSynchronize();
        err = hipGetLastError();
        /*if( err != hipSuccess )
        {
            printf("Errors found :\n %s", hipGetErrorString(err));
        }*/

        for( std::size_t i = 0 ; i < streamsNumber ; ++i ){
            hipMemcpyAsync( out.data() + i * size_data_out, data_out_streams[i], size_data_out, hipMemcpyDeviceToHost, streams[i]);
        }
        
        // On récupère le temps d'exécution
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Execution time : %f\n",milliseconds);

        cv::imwrite( "outCudaV3.jpg", image_out);


        for( std::size_t i = 0 ; i < streamsNumber ; ++i ){
            hipStreamDestroy(streams[i]);
            hipFree(data_in_streams[i]);
            hipFree(data_out_streams[i]);
        }
    }

    return 0;
}

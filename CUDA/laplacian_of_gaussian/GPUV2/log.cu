#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void laplacian_of_gaussian(unsigned char * data_rgb, unsigned char * const data_out, std::size_t rows, std::size_t cols)
{
    auto i = blockIdx.x * (blockDim.x - 4)+ threadIdx.x; //car besoin de 4 pxl pour calculer
    auto j = blockIdx.y * (blockDim.y - 4) + threadIdx.y;

    auto gray_i = threadIdx.x;
    auto gray_j = threadIdx.y;

    extern __shared__ unsigned char data_gray[];

    auto cols_gray = blockDim.x;


    data_gray[ gray_j * cols_gray + gray_i ] = ( 
            307 * data_rgb[ 3 * (j * cols + i) ]
        +   604 * data_rgb[ 3 * (j * cols + i) + 1 ]
        +   113 * data_rgb[ 3 * (j * cols + i) + 2 ]
    ) / 1024;
        

    __syncthreads();


    if( gray_i > 1 && gray_i < (cols_gray - 2) && gray_j > 1 && gray_j < (blockDim.y - 2))
    {
        // Tous les pixels que l'on multiplie par 16
        auto result = data_gray[(gray_j * cols_gray + gray_i)] * 16

        // Tous les pixels que l'on multiplie par -2
        + ( data_gray[((gray_j-1) * cols_gray + gray_i)] + data_gray[((gray_j+1) * cols_gray + gray_i)] + data_gray[(gray_j * cols_gray + (gray_i-1))] + data_gray[(gray_j * cols_gray + (gray_i+1))] ) * -2

        // Tous les pixels que l'on multiplie par -1
        + ( data_gray[((gray_j-2) * cols_gray + gray_i)] + data_gray[((gray_j+2) * cols_gray + gray_i)] + data_gray[(gray_j * cols_gray + (gray_i-2))] + data_gray[(gray_j * cols_gray + (gray_i+2))] 
            + data_gray[((gray_j-1) * cols_gray + (gray_i-1))] + data_gray[((gray_j-1) * cols_gray + (gray_i+1))] + data_gray[((gray_j+1) * cols_gray + (gray_i-1))] + data_gray[((gray_j+1) * cols_gray + (gray_i+1))] ) * -1;


        result = result * result;
        result = result > 255*255 ? result = 255*255 : result;
        data_out[ j * cols + i ] = sqrt((float) result);
    }
}

int main(int argc, char** argv)
{
    //printf("Number of argument : %d\n", argc);

    if(argc >= 2){

        int threadSize = 32;

        if(argc == 3){
            threadSize = atoi(argv[2]);
        }

        // Mesure de temps
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        //std::cout << "Création du timer faite" << std::endl;

        // Récupère l'image
        cv::Mat image_in = cv::imread(argv[1], cv::IMREAD_UNCHANGED);
        // Récupère les informations des pixels
        auto data_rgb = image_in.data;
        auto rows = image_in.rows;
        auto cols = image_in.cols;

	
        std::cout << "rows = " << rows << " columns = " << cols << std::endl;

        // On crée les informations de sorties 
        std::vector<unsigned char> out(rows * cols); 
        // On crée l'image de sortie
        cv::Mat image_out(rows, cols, CV_8UC1, out.data());

        //std::cout << "Image et données de sortie initialisées" << std::endl;

        // On copie l'image d'entrée sur le device
        unsigned char * data_rgb_device;
        // On crée une copie des informations de sortie sur le device
        unsigned char* data_out_device;

        hipMalloc(&data_rgb_device, 3 * rows * cols);
        hipMalloc(&data_out_device, rows * cols);

        //std::cout << "Image sur le device allouée" << std::endl;

        //std::cout << "Données de sortie sur le device allouées" << std::endl;

        hipMemcpy(data_rgb_device, data_rgb,  3 * rows * cols, hipMemcpyHostToDevice );
                                                                                    
        //std::cout << "Image d'entrée mise sur le device" << std::endl;

        dim3 threads(threadSize, threadSize );
        dim3 blocks(( cols -1 ) / (threads.x-4) + 1 , ( rows - 1) / (threads.y-4) + 1);

        std::cout << "Nombre de threads = " << threads.x << "  " << threads.y << std::endl;
        std::cout << "Nombre de blocks = " << blocks.x << "  " << blocks.y << std::endl;

        // Lancement du timer
        hipEventRecord(start);

        //std::cout << "Lancement du timer" << std::endl;
        
        // lancement du programme
        laplacian_of_gaussian<<< blocks , threads , threadSize * threadSize>>>(data_rgb_device, data_out_device, rows, cols);

        // On arrête le timer
        hipEventRecord(stop);

        //std::cout << "Fin du timer" << std::endl;

        hipDeviceSynchronize();
        /*auto err = hipGetLastError();
        if( err != hipSuccess )
        {
            printf("Errors found :\n %s", hipGetErrorString(err));
        }*/

        // On copie les informations de sortie du device vers le host
        hipMemcpy(out.data(), data_out_device, rows * cols, hipMemcpyDeviceToHost );
        
        // On récupère le temps d'exécution
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Execution time : %f\n",milliseconds);

        cv::imwrite( "outCudaV2.jpg", image_out);

        // On libère l'espace sur le device
        hipFree(data_rgb_device);
        hipFree(data_out_device);
    }

    return 0;
}

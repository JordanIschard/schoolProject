#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>


__global__ void grayscale(unsigned char * data_rgb, unsigned char * data_gray, std::size_t rows, std::size_t cols)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if( i < cols && j < rows )
    {
        data_gray[ j * cols + i ] = ( 
                307 * data_rgb[ 3 * (j * cols + i) ]
            +   604 * data_rgb[ 3 * (j * cols + i) + 1 ]
            +   113 * data_rgb[ 3 * (j * cols + i) + 2 ]
        ) / 1024;
    }
}

__global__ void simple_box_blur(unsigned char const * const data_gray, unsigned char * const data_out, std::size_t rows, std::size_t cols)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if( i >= 1 && i < (cols - 1) && j >= 1 && j < (rows - 1))
    {
		auto result = (data_gray[(j * cols + i)] 
		+ data_gray[((j-1) * cols + i)] + data_gray[((j+1) * cols + i)]
		+ data_gray[(j * cols + (i-1))] + data_gray[(j * cols + (i+1))] 	
		+ data_gray[((j-1) * cols +(i-1))] + data_gray[((j-1) * cols +(i+1))]
		+ data_gray[((j+1) * cols +(i-1))] + data_gray[((j+1) * cols +(i+1))]	
		) /9;

        result = result * result;
        result = result > 255*255 ? result = 255*255 : result;
        data_out[ j * cols + i ] = sqrt((float) result);
    }
}

__global__ void edge_detection(unsigned char const * const data_in, unsigned char * const data_out, std::size_t rows, std::size_t cols)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if( i >= 1 && i < (cols - 1) && j >= 1 && j < (rows - 1))
    {
		auto result = data_in[(j * cols + i)] *8 +(
		+ data_in[((j-1) * cols + i)] + data_in[((j+1) * cols + i)]
		+ data_in[(j * cols + (i-1))] + data_in[(j * cols + (i+1))] 	
		+ data_in[((j-1) * cols +(i-1))] + data_in[((j-1) * cols +(i+1))]
		+ data_in[((j+1) * cols +(i-1))] + data_in[((j+1) * cols +(i+1))]	
		) *(-1);

        result = result * result;
        result = result > 255*255 ? result = 255*255 : result;
        data_out[ j * cols + i ] = sqrt((float) result);
    }
}

int main(int argc, char** argv)
{
    //printf("Number of argument : %d\n", argc);

    if(argc >= 2){

        int threadSize = 32;

        if(argc == 3){
            threadSize = atoi(argv[2]);
        }

        // Mesure de temps
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Récupère l'image
        cv::Mat image_in = cv::imread(argv[1], cv::IMREAD_UNCHANGED);

        // Récupère les informations des pixels
        auto data_rgb = image_in.data;
        auto rows = image_in.rows;
        auto cols = image_in.cols;

	
        std::cout << "rows = " << rows << " columns = " << cols << std::endl;

        // On crée les informations de sorties 
        std::vector<unsigned char> out(rows * cols); 
        // On crée l'image de sortie
        cv::Mat image_out(rows, cols, CV_8UC1, out.data());

        // On copie l'image d'entrée sur le device
        unsigned char * data_rgb_device;
        unsigned char * data_gray_device;
        // On crée une copie des informations de sortie sur le device
        unsigned char* data_out_device;
        unsigned char * data_edge_device;

        hipMalloc(&data_rgb_device, 3 * rows * cols);
        hipMalloc(&data_gray_device, rows * cols);
        hipMalloc(&data_out_device, rows * cols);
        hipMalloc(&data_edge_device, rows * cols);

        hipMemcpy(data_rgb_device, data_rgb,  3 * rows * cols, hipMemcpyHostToDevice );

        dim3 threads(threadSize, threadSize );
        dim3 blocks(( cols -1 ) / threads.x + 1 , ( rows - 1) / threads.y + 1);

        std::cout << "Nombre de threads = " << threads.x << "  " << threads.y << std::endl;
        std::cout << "Nombre de blocks = " << blocks.x << "  " << blocks.y << std::endl;

        // Lancement du timer
        hipEventRecord(start);
        
        grayscale<<< blocks , threads >>>(data_rgb_device, data_gray_device, rows, cols);

        // lancement du programme
		edge_detection<<< blocks , threads >>>(data_gray_device, data_edge_device, rows, cols);
        simple_box_blur<<< blocks , threads >>>(data_edge_device, data_out_device, rows, cols);

        // On arrête le timer
        hipEventRecord(stop);

        hipDeviceSynchronize();
        /*auto err = hipGetLastError();
        if( err != hipSuccess )
        {
            printf("Errors found :\n %s", hipGetErrorString(err));
        }*/

        // On copie les informations de sortie du device vers le host
        hipMemcpy(out.data(), data_out_device, rows * cols, hipMemcpyDeviceToHost );
        
        // On récupère le temps d'exécution
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Execution time : %f\n",milliseconds);

        cv::imwrite( "outSbbV2.jpg", image_out);

        // On libère l'espace sur le device
        hipFree(data_rgb_device);
        hipFree(data_gray_device);
        hipFree(data_out_device);
		hipFree(data_edge_device);
    }

    return 0;
}
